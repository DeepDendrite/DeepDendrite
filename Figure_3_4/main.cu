#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <string.h>
#include <>
#include "solver_kernels.cuh"

using namespace std;

template <typename T> void node_permute(T *arr, int len, int* order)
{
    T *arr_tmp = new T[len];
    for (int i = 0; i < len; i++)
    {
        arr_tmp[i] = arr[i];
    }

    for (int i = 0; i < len; i++)
    {
        arr[order[i]] = arr_tmp[i];
    }
    delete[] arr_tmp;
}

void index_permute(int *arr, int len, int *order)
{
    int *arr_tmp = new int[len];
    for (int i = 0; i < len; i++)
    {
        arr_tmp[i] = arr[i];
    }

    for (int i = 0; i < len; i++)
    {
        arr[order[i]] = order[arr_tmp[i]];
    }
    delete[] arr_tmp;
}

int* inverse_permute(int *order, int len)
{
    int *order_inv = new int[len];
    for (int i = 0; i < len; i++)
    {
        order_inv[order[i]] = i;
    }
    return order_inv;
}

void invert_permute(int *order, int len)
{
    int *order_inv = inverse_permute(order, len);
    for (int i = 0; i < len; i++)
    {
        order[i] = order_inv[i];
    }
    delete[] order_inv;
}

int main(int argc,char* argv[])
{
	//int ncell = 131072;
	//int len = 4333072;
	//int nstride = 42;
    FILE *profile;
    char profile_name[300], cell_data_filename[300], result_filename[300];
    sprintf(profile_name, "%s/%s", argv[1], "cell_profile");
    sprintf(cell_data_filename, "%s/%s", argv[1], "cell_data");
    sprintf(result_filename, "%s/%s", argv[1], "result_out.txt");
    profile = fopen(profile_name,"r");
	int ncell;
    int len, threads_num, norder;
    int nstride;
    int solver_type;
    int device_id;
    solver_type = atoi(argv[argc - 1]);
    device_id = atoi(argv[argc - 2]);
    hipSetDevice(device_id);
    /*if (argc == 5)
        solver_type = atoi(argv[4]);
    if (argc == 6)
        solver_type = atoi(argv[5]);*/

    fscanf(profile,"%d",&ncell);
    fscanf(profile,"%d",&len);
    
    if (solver_type == 0)
    {
        fscanf(profile,"%d",&nstride);
    }

    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        fscanf(profile, "%d%d", &threads_num, &norder);
    }
    fclose(profile);
    unsigned int threads_per_block = 128;
	unsigned int block_num = (ncell + threads_per_block - 1) / threads_per_block;

	double *a_host, *b_host, *d_host, *rhs_host;
	int *parent_host, *stride_host, *firstnode_host, *lastnode_host, *cellsize_host;
    int *min_level_host, *max_level_host, *prev_host, *next_host;

	double* vec_a, *vec_b, *vec_d, *vec_rhs;
	int* parent_index, *stride, *firstnode, *lastnode, *cellsize;
    int *min_level_per_thread, *max_level_per_thread, *prev_node, *next_node;
	float *rhs_gt, *d_gt;

	FILE* fp;
	fp = fopen(cell_data_filename, "rb");

	hipMalloc((void**)&vec_a, sizeof(double)*len);
	hipMalloc((void**)&vec_b, sizeof(double)*len);
	hipMalloc((void**)&vec_d, sizeof(double)*len);
	hipMalloc((void**)&vec_rhs, sizeof(double)*len);
	hipMalloc((void**)&parent_index, sizeof(int)*len);

	if (solver_type == 0) 
    {
        hipMalloc((void**)&stride, sizeof(int)*(nstride+1));
	    hipMalloc((void**)&firstnode, sizeof(int)*ncell);
	    hipMalloc((void**)&lastnode, sizeof(int)*ncell);
	    hipMalloc((void**)&cellsize, sizeof(int)*ncell);
    }
    else if ( solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        hipMalloc((void**)&cellsize, sizeof(int) * threads_num);
        hipMalloc((void**)&firstnode, sizeof(int) * threads_num);
        hipMalloc((void**)&lastnode, sizeof(int) * threads_num);
        hipMalloc((void**)&max_level_per_thread, sizeof(int) * threads_num);
        hipMalloc((void**)&min_level_per_thread, sizeof(int) * threads_num);
        hipMalloc((void**)&prev_node, sizeof(int) * len);
        hipMalloc((void**)&next_node, sizeof(int) * len);

    }
    

	a_host = (double*)malloc(sizeof(double)*len);
	b_host = (double*)malloc(sizeof(double)*len);
	d_host = (double*)malloc(sizeof(double)*len);
	rhs_host = (double*)malloc(sizeof(double)*len);
	parent_host = (int*)malloc(sizeof(int)*len);
    if (solver_type == 0)	
    {
        stride_host = (int*)malloc(sizeof(int)*(nstride+1));
        firstnode_host = (int*)malloc(sizeof(int)*ncell);
        lastnode_host = (int*)malloc(sizeof(int)*ncell);
        cellsize_host = (int*)malloc(sizeof(int)*ncell);
    }
    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        cellsize_host = (int*)malloc(sizeof(int) * threads_num);
        next_host = (int*)malloc(sizeof(int) * len);
        prev_host = (int*)malloc(sizeof(int) * len);
        firstnode_host = (int*)malloc(sizeof(int) * threads_num);
        lastnode_host = (int*)malloc(sizeof(int) * threads_num);
        max_level_host = (int*)malloc(sizeof(int) * threads_num);
        min_level_host = (int*)malloc(sizeof(int) * threads_num);
    }
    

	fread(a_host, sizeof(double), len, fp);
	fread(b_host, sizeof(double), len, fp);
	fread(d_host, sizeof(double), len, fp);
	fread(rhs_host, sizeof(double), len, fp);
	fread(parent_host, sizeof(int), len, fp);
    if (solver_type == 0) 
    {
        fread(stride_host, sizeof(int), nstride+1, fp);
        fread(firstnode_host, sizeof(int), ncell, fp);
        fread(lastnode_host, sizeof(int), ncell, fp);
        fread(cellsize_host, sizeof(int), ncell, fp);
    }
    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        fread(prev_host, sizeof(int), len, fp);
        fread(next_host, sizeof(int), len, fp);
        fread(max_level_host, sizeof(int), threads_num, fp);
        fread(min_level_host, sizeof(int), threads_num, fp);
        fread(firstnode_host, sizeof(int), threads_num, fp);
        fread(lastnode_host, sizeof(int), threads_num, fp);
        
    }
	fclose(fp);
	
    // We use kernel16 to test the run time without 
    // GPU memory optimization. The corresponding
    // data read from the files are in optimized
    // order. So if we want to test the run time 
    // without GPU memory optimization, we should
    // first transform the data from optimized order
    // to the original order.
    int *order3 = NULL;
    if (solver_type == 16)
    {
        char order_filename[300];
        sprintf(order_filename, "%s/%s", argv[1], "order3");
        printf("%s\n", order_filename);
        fp = fopen(order_filename, "rb");
        order3 = new int[len];
        fread(order3, sizeof(int), len, fp);
        fclose(fp);

        //permute all arrays from optimized order to original order
        invert_permute(order3, len); // now order3 map from order3 to original
        node_permute(a_host, len, order3);
        node_permute(b_host, len, order3);
        node_permute(d_host, len, order3);
        node_permute(rhs_host, len, order3);
        index_permute(parent_host, len, order3);
        index_permute(prev_host, len, order3);
        index_permute(next_host, len, order3);

        for (int i = 0; i < threads_num; i++)
        {
            firstnode_host[i] = order3[firstnode_host[i]];
            lastnode_host[i] = order3[lastnode_host[i]];
        }
    }

	hipMemcpy(vec_a, a_host, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(vec_b, b_host, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(vec_d, d_host, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(vec_rhs, rhs_host, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(parent_index, parent_host, sizeof(int)*len, hipMemcpyHostToDevice);

    if (solver_type == 0)
    {
        hipMemcpy(stride, stride_host, sizeof(int)*(nstride+1), hipMemcpyHostToDevice);
        hipMemcpy(firstnode, firstnode_host, sizeof(int)*ncell, hipMemcpyHostToDevice);
        hipMemcpy(lastnode, lastnode_host, sizeof(int)*ncell, hipMemcpyHostToDevice);
        hipMemcpy(cellsize, cellsize_host, sizeof(int)*ncell, hipMemcpyHostToDevice);
    }
    
    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        hipMemcpy(prev_node, prev_host, sizeof(int) * len, hipMemcpyHostToDevice);
        hipMemcpy(next_node, next_host, sizeof(int) * len, hipMemcpyHostToDevice);
        hipMemcpy(firstnode, firstnode_host, sizeof(int) * threads_num, hipMemcpyHostToDevice);
        hipMemcpy(lastnode, lastnode_host, sizeof(int) * threads_num, hipMemcpyHostToDevice);
        hipMemcpy(max_level_per_thread, max_level_host, sizeof(int) * threads_num, hipMemcpyHostToDevice);
        hipMemcpy(min_level_per_thread, min_level_host, sizeof(int) * threads_num, hipMemcpyHostToDevice);
    }
    
	//hipMemcpy(children_node, children_node_host, sizeof(int)*ncell, hipMemcpyHostToDevice);
    //hipMemcpy(map_a, map_a_host, sizeof(int)*len, hipMemcpyHostToDevice);
	//hipMemcpy(map_b, map_b_host, sizeof(int)*len, hipMemcpyHostToDevice);

    printf("ncell:%d len:%d\n", ncell, len);
  if (solver_type == 0)
  {
	  solve_kernel0 <<< block_num, threads_per_block >>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, nstride, stride, firstnode, lastnode, cellsize, ncell);
      //get_single_cell(parent_host, ncell, len, cellsize_host, stride_host, nstride, firstnode_host);
  }
  
    else if (solver_type == 16)
    {
        threads_per_block = 128;
        int branch_per_cell = threads_num / ncell;
        int ncell_per_warp = 32 / branch_per_cell;
        block_num = (threads_num + threads_per_block - 1) / threads_per_block;
        kernel16<<<block_num, threads_per_block>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, 
                prev_node, next_node, max_level_per_thread, min_level_per_thread, firstnode, lastnode, norder, ncell, threads_num, 0);

    }
    else if (solver_type == 17 || solver_type == 18)
    {
        int *stride_host = NULL, *stride = NULL;
        int nwarp;
        if (threads_num % 32 == 0)
            nwarp = threads_num / 32;
        else
            nwarp = threads_num / 32 + 1;

        stride_host = (int*)malloc(sizeof(int) * nwarp * (norder + 1));
        char stride_filename[300];
        sprintf(stride_filename, "%s/%s", argv[1], "stride_array");
        FILE *stride_file = fopen(stride_filename, "rb");
        fread(stride_host, sizeof(int), nwarp * (norder + 1), stride_file);
        fclose(stride_file);

        hipMalloc((void**)&stride, sizeof(int) * nwarp * (norder + 1));
        hipMemcpy(stride, stride_host, sizeof(int) * nwarp * (norder + 1), hipMemcpyHostToDevice);

        threads_per_block = 32;
        int branch_per_cell = threads_num / ncell;
        int ncell_per_warp = 32 / branch_per_cell;
        block_num = (threads_num + threads_per_block - 1) / threads_per_block;

        //printf("shared memeory:%d B\n", sizeof(int) * threads_per_block / 32 * (norder + 1));
        //kernel17<<<block_num, threads_per_block, sizeof(int) * threads_per_block / 32 * (norder + 1)>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, 
        //        stride, max_level_per_thread, min_level_per_thread, firstnode, lastnode, norder, ncell, threads_num, 0);
        if (solver_type == 17)
        {
            hipFuncSetAttribute(reinterpret_cast<const void*>(kernel17), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
            kernel17<<<block_num, threads_per_block>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, stride, max_level_per_thread, min_level_per_thread, firstnode, lastnode, norder, ncell, threads_num);
        }

        if (solver_type == 18)
        {
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("%s\n", hipGetErrorString(err));
            else
                printf("success\n");
            //hipFuncSetAttribute(reinterpret_cast<const void*>(kernel18), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
            hipDeviceSynchronize();
            printf("block_num:%d %d nthread:%d\n", block_num, block_num / 2, threads_num);
            kernel18<<<block_num / 2, threads_per_block>>>(vec_a, vec_b, vec_d, vec_rhs, parent_index, stride, max_level_per_thread, min_level_per_thread, firstnode, lastnode, norder, ncell, threads_num);
            err = hipGetLastError();
            if (err != hipSuccess)
                printf("%s\n", hipGetErrorString(err));
            else
                printf("success\n");
        }

        hipFree(stride);
        free(stride_host);
    }
	//hipMemcpy(count_host,condition_count,sizeof(int)*2,hipMemcpyDeviceToHost);
    //printf("%d %d\n",count_host[0],count_host[1]);

    if (solver_type != 12)
    {
        hipMemcpy(rhs_host, vec_rhs, sizeof(double)*len, hipMemcpyDeviceToHost);
        hipMemcpy(d_host, vec_d, sizeof(double)*len, hipMemcpyDeviceToHost);
    }
	d_gt = (float*)malloc(sizeof(float)*len);
	rhs_gt = (float*)malloc(sizeof(float)*len);
	double err = 0;
	FILE* result_gt;
	result_gt = fopen(result_filename, "r");
	for (int i = 0; i < len; i++)
	{
		fscanf(result_gt, "%f", &rhs_gt[i]);
	}
	for (int i = 0; i < len; i++)
	{
		fscanf(result_gt, "%f", &d_gt[i]);
	}
    if (solver_type == 16 && order3)
    {
        node_permute(rhs_gt, len, order3);
        node_permute(d_gt, len, order3);
    }
	for (int i = 0; i < len; i++)
	{
        if (rhs_host[i]!=rhs_host[i])
        {
            printf("compute error:%d\n", i);
            break;
        }
        if (rhs_gt[i]!=rhs_gt[i])
            printf("gt error:%d\n", i);
		err += (rhs_gt[i] - rhs_host[i])*(rhs_gt[i] - rhs_host[i]);
	}
	printf("error: %f\n", err);
    fclose(result_gt);
	free(a_host);
    free(d_gt);
    free(rhs_gt);
	free(b_host);
	free(d_host);
	free(rhs_host);
	free(parent_host);
	free(firstnode_host);
	free(lastnode_host);
    if (order3)
    {
        delete[] order3;
        order3 = NULL;
    }
    if (solver_type == 0)
    {
	    free(cellsize_host);
	    free(stride_host);
    }
    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        free(next_host);
        free(prev_host);
        free(max_level_host);
        free(min_level_host);
    }
	
	hipFree(vec_a);
	hipFree(vec_b);
	hipFree(vec_d);
	hipFree(vec_rhs);
	hipFree(parent_index);
	hipFree(firstnode);
	hipFree(lastnode);
    
    if (solver_type == 0)
    {
        hipFree(stride);
	    hipFree(cellsize);
    }
    else if (solver_type == 17 || solver_type == 16 || solver_type == 18)
    {
        //hipFree(next_node);
        //hipFree(prev_node);
        hipFree(max_level_per_thread);
        hipFree(min_level_per_thread);
    }

	return 0;
}











