#include "hip/hip_runtime.h"
#include <hipsparse.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <stdio.h>
#include "solver_kernels.cuh"



__global__ void kernel16(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell;
    double p;
    double temp, reg_rhs;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                //if (i < 0 || i >= 407168 || ip < 0 || ip >= 407168)
                //    printf("error icell:%d lastnode:%d i:%d ip:%d stride:%d\n", icell, lastnode[icell], i, ip, stride[istride]);
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                //temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                //atomicAdd(vec_d + ip, temp);
		atomicAdd(vec_d + ip, -p * vec_b[i]);
                //temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);
                //atomicAdd(vec_rhs + ip, temp);
		atomicAdd(vec_rhs + ip, -p * vec_rhs[i]);

                i = prev_node[i];
            }
            //__syncthreads();
        }


        ncell_per_warp = 32 / (nthread / ncell);
        i=firstnode[tid];
        if (tid % 32 < ncell_per_warp)
        {
            icell = tid / 32 * ncell_per_warp + tid % 32;
            //if (icell < 4)
            //    printf("tid:%d icell:%d\n", tid, icell);
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
                
                reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                vec_rhs[i] = reg_rhs / vec_d[i];
                //vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] /= vec_d[i];
                i = next_node[i];    
            }
            //__syncthreads();
            
        }
        tid += thread_stride;
    }
}


/*__global__ void kernel16(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, bool* reg_avai, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockidx.x * blockdim.x + threadidx.x;
    unsigned int thread_stride = blockdim.x * griddim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell;
    double p;
    double temp, reg_rhs, reg_d;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);
    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];

                
                //if (i < 0 || i >= 407168 || ip < 0 || ip >= 407168)
                //    printf("error icell:%d lastnode:%d i:%d ip:%d stride:%d\n", icell, lastnode[icell], i, ip, stride[istride]);
                p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                temp = -p * vec_b[i];
                atomicdoubleadd(vec_d + ip, temp);
                temp = -p * vec_rhs[i];
                atomicdoubleadd(vec_rhs + ip, temp);

                i = prev_node[i];
            }
            //__syncthreads();
        }


        ncell_per_warp = 32 / (nthread / ncell);
        i=firstnode[tid];
        if (tid % 32 < ncell_per_warp)
        {
            icell = tid / 32 * ncell_per_warp + tid % 32;
            //if (icell < 4)
            //    printf("tid:%d icell:%d\n", tid, icell);
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }

        //use_reg = 0;
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip = parent_index[i];           
                reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                vec_rhs[i] = reg_rhs / vec_d[i];
                //vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] /= vec_d[i];
                i = next_node[i];
            }
            //__syncthreads();
            
        }
        tid += thread_stride;
    }
}*/

//kernel16 with opt
/*__global__ void kernel16(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* prev_node, int* next_node, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, bool* reg_avai, int norder, int ncell, int nthread, double t)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thread_stride = blockDim.x * gridDim.x;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip, ncell_per_warp, icell;
    bool use_reg = false, use_reg2 = false;
    double p;
    double temp, reg_rhs, reg_d;
    int max_order = max_order_per_thread[tid];
    int min_order = min_order_per_thread[tid];
    //if ((icell == 0 || icell == 636) )
    //    printf("%d %d\n", icell, iter);

    extern __shared__ bool s_reg_avai[];
    for (i = 0; i <= norder; i++)
    {
        s_reg_avai[i * blockDim.x + threadIdx.x] = reg_avai[i * 128 + tid];
    }

    while (tid < nthread)
    {
        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            
            if(iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip=parent_index[i];
    
                if (!use_reg)
                {
                    p = vec_a[i] / vec_d[i];
                }
                else
                {
                    p = vec_a[i] / reg_d;
                }
                use_reg2 = s_reg_avai[(iorder - 1) * blockDim.x + threadIdx.x];
                //s_reg_avai[iorder * blockDim.x + threadIdx.x] = use_reg;
                temp = -p * vec_b[i];
                if (use_reg2)
                {
                    reg_d = vec_d[ip] + temp;
                    vec_d[ip] = reg_d;
                    if (use_reg)
                    {
                        reg_rhs = vec_rhs[ip] - p * reg_rhs;
                    }
                    else
                    {
                        reg_rhs = vec_rhs[ip] - p * vec_rhs[i];
                    }
                    vec_rhs[ip] = reg_rhs;
                }
                else
                {
                    atomicDoubleAdd(vec_d + ip, temp);
                    if (use_reg)
                    {
                        temp = -p * reg_rhs;
                    }
                    else
                    {
                        temp = -p * vec_rhs[i];
                    }
                    atomicDoubleAdd(vec_rhs + ip, temp);
                }
                
                //p = vec_a[i]/vec_d[i];
                //vec_d[ip] -= p*vec_b[i];
                //vec_rhs[ip] -= p*vec_rhs[i];
                //temp = -p * vec_b[i];
                //atomicDoubleAdd(vec_d + ip, temp);
                //temp = -p * vec_rhs[i];
                //atomicDoubleAdd(vec_rhs + ip, temp);

                use_reg = use_reg2;
                i = prev_node[i];
            }
        }


        ncell_per_warp = 32 / (nthread / ncell);
        i=firstnode[tid];
        if (tid % 32 < ncell_per_warp)
        {
            icell = tid / 32 * ncell_per_warp + tid % 32;
            //if (icell < 4)
            //    printf("tid:%d icell:%d\n", tid, icell);
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }

        //use_reg = 0;
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && iorder <= max_order && i > -1)
            {
                ip = parent_index[i];           
                if (s_reg_avai[(iorder - 1) * blockDim.x + threadIdx.x])
                //{
                //    temp = vec_rhs[i] - vec_b[i] * reg_rhs;
                //}
                //else
                //{
                //    temp = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                //}
                //vec_rhs[i] = reg_rhs = temp / vec_d[i];
                reg_rhs = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                vec_rhs[i] = reg_rhs / vec_d[i];
                //vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
                //vec_rhs[i] /= vec_d[i];
                i = next_node[i];
            }
            //__syncthreads();
            
        }
        tid += thread_stride;
    }
}*/




