#include "hip/hip_runtime.h"
#include "solver_kernels.cuh"

__global__ void solve_kernel0(double* vec_a,
	double* vec_b,
	double* vec_d,
	double* vec_rhs,
	int* parent_index,
	int nstride, int* stride, int* firstnode, int* lastnode, int* cellsize, int ncell)
{
	unsigned int tid;
	unsigned int thread_stride;
	int i, istride, ip, icell, icellsize;
	double p;
	thread_stride = blockDim.x * gridDim.x;
	tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid<ncell)
	{
		//triang_kernel(vec_a, vec_b, vec_d, vec_rhs, parent_index, tid, cellsize[tid], nstride, s_stride, lastnode, condition_count);
		//triang_kernel2(vec_a, vec_b, vec_d, vec_rhs, parent_index, tid, cellsize[tid], nstride, stride, lastnode, condition_count);
		//bksub_kernel(vec_a, vec_b, vec_d, vec_rhs, parent_index, tid, cellsize[tid], nstride, s_stride, firstnode);
		//elimination(vec_a, vec_b, vec_d, vec_rhs, parent_index, tid, cellsize[tid], nstride, stride, firstnode, lastnode);
		//elimination2(vec_a, vec_b, vec_d, vec_rhs, parent_index, cellsize[tid], tid, node_start_index, children_node, nstride, firstnode,lastnode, stride);

		//triang
		icell = tid;
		icellsize = cellsize[icell];
		i = lastnode[icell];
		for (istride = nstride - 1; istride >= 0; --istride)
		{
			if (istride < icellsize)
			{
				ip = parent_index[i];
				p = vec_a[i] / vec_d[i];
				vec_d[ip] -= p * vec_b[i];
				vec_rhs[ip] -= p * vec_rhs[i];
				i -= stride[istride];
			}
		}

		//bksub
		i = firstnode[icell];
		vec_rhs[icell] /= vec_d[icell];
		for (istride = 0; istride < icellsize; ++istride)
		{
			ip = parent_index[i];
			vec_rhs[i] -= vec_b[i] * vec_rhs[ip];
			vec_rhs[i] /= vec_d[i];
			i += stride[istride + 1];
		}

		tid += thread_stride;
	}
}
