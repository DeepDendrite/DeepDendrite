#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "solver_kernels.cuh"


__global__ void kernel18(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* stride, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int norder, int ncell, int nthread)
{
    unsigned int tid;
    int i, iorder, ip, next_i, next_ip;
    double p;
    int min_order;
    double cur_a, cur_b, next_a, next_b;
    //start_pos = blockIdx.x * (blockDim.x / 32) * (norder + 1);
    tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nthread)
    {
        int start_pos = (tid >> 5) * (norder + 1) - 1;
        //start_pos = tid / 32 * (norder + 1);

        //max_order = max_order_per_thread[tid];
        min_order = min_order_per_thread[tid];

        i = lastnode[tid];
        next_a = vec_a[i];
        next_b = vec_b[i];
        next_ip = parent_index[i];

        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && i > -1)
            {
                next_i = i - stride[start_pos + iorder];
                cur_a = next_a;
                cur_b = next_b;
                ip = next_ip;
                if (next_i > -1)
                {
                    next_a = vec_a[next_i];
                    next_b = vec_b[next_i];
                    next_ip = parent_index[next_i];
                }

                p = cur_a / vec_d[i];
                atomicAdd(vec_d + ip, -p * cur_b);
                atomicAdd(vec_rhs + ip, -p * vec_rhs[i]);
                i = next_i;
            }
        }


        int ncell_each_warp = 32 / (nthread / ncell);
        int icell = (tid >> 5) * (32 / (nthread / ncell)) + (tid & 31);
        i=firstnode[tid];
        //next_i=firstnode[tid];
        if ((tid & 31) < ncell_each_warp) 
        {
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }

        next_b = vec_b[i];
        next_ip = parent_index[i];
        start_pos = (tid >> 5) * (norder + 1);
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && i > -1)
            {
                next_i = i + stride[start_pos + iorder];
                ip = next_ip;
                next_ip = parent_index[next_i];
                cur_b = next_b;
                next_b = vec_b[next_i];
                //ip = parent_index[i];
                //p = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                p = vec_rhs[i] - cur_b * vec_rhs[ip];
                vec_rhs[i] = p / vec_d[i];
                i = next_i;
                //i += stride[start_pos + iorder];
            }
        }

    }

    tid += nthread / 2;
    if (tid < nthread)
    {
        int start_pos = (tid >> 5) * (norder + 1) - 1;
        //start_pos = tid / 32 * (norder + 1);

        //max_order = max_order_per_thread[tid];
        min_order = min_order_per_thread[tid];

        i = lastnode[tid];
        next_a = vec_a[i];
        next_b = vec_b[i];
        next_ip = parent_index[i];

        for(iorder = norder; iorder >= 0; iorder--)
        {
            if(iorder >= min_order && i > -1)
            {
                next_i = i - stride[start_pos + iorder];
                cur_a = next_a;
                cur_b = next_b;
                ip = next_ip;
                if (next_i > -1)
                {
                    next_a = vec_a[next_i];
                    next_b = vec_b[next_i];
                    next_ip = parent_index[next_i];
                }

                p = cur_a / vec_d[i];
                atomicAdd(vec_d + ip, -p * cur_b);
                atomicAdd(vec_rhs + ip, -p * vec_rhs[i]);
                i = next_i;
            }
        }


        int ncell_each_warp = 32 / (nthread / ncell);
        int icell = (tid >> 5) * (32 / (nthread / ncell)) + (tid & 31);
        i=firstnode[tid];
        //next_i=firstnode[tid];
        if ((tid & 31) < ncell_each_warp) 
        {
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }

        next_b = vec_b[i];
        next_ip = parent_index[i];
        start_pos = (tid >> 5) * (norder + 1);
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && i > -1)
            {
                next_i = i + stride[start_pos + iorder];
                ip = next_ip;
                next_ip = parent_index[next_i];
                cur_b = next_b;
                next_b = vec_b[next_i];
                //ip = parent_index[i];
                //p = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                p = vec_rhs[i] - cur_b * vec_rhs[ip];
                vec_rhs[i] = p / vec_d[i];
                i = next_i;
                //i += stride[start_pos + iorder];
            }
        }

    }
    
}







