#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "solver_kernels.cuh"


__global__ void kernel17(double* vec_a, double* vec_b, double* vec_d, double* vec_rhs, int* parent_index,
                        int* stride, int* max_order_per_thread, int* min_order_per_thread, 
                        int* firstnode, int* lastnode, int norder, int ncell, int nthread)
{
    unsigned int tid;
    //unsigned int icellsize = cellsize[tid];
    int i, iorder, ip;
    //int icell;
    double p;
    int min_order;
    
    tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < nthread)
    {
        //start_pos = threadIdx.x / 32 * (norder + 1);
        //start_pos = tid / 32 * (norder + 1);

        //max_order = max_order_per_thread[tid];
        min_order = min_order_per_thread[tid];

        i = lastnode[tid];
        for(iorder = norder; iorder >= 0; iorder--)
        {
            
            if(iorder >= min_order && i > -1)
            {
                ip = parent_index[i];
                p = vec_a[i] / vec_d[i];
                atomicAdd(vec_d + ip, -p * vec_b[i]);
                atomicAdd(vec_rhs + ip, -p * vec_rhs[i]);

                i -= stride[(tid >> 5) * (norder + 1) + iorder - 1];
            }
            //__syncthreads();
        }


        //ncell_per_warp = 32 / (nthread / ncell);
        int ncell_each_warp = 32 / (nthread / ncell);
        int icell = (tid >> 5) * (32 / (nthread / ncell)) + (tid & 31);
        i=firstnode[tid];
        if ((tid & 31) < ncell_each_warp) 
        {
            if (icell < ncell)
                vec_rhs[icell] /= vec_d[icell];
        }
        for(iorder = 1; iorder <= norder; ++iorder)
        {
            if (iorder >= min_order && i > -1)
            {
                ip = parent_index[i];

                p = vec_rhs[i] - vec_b[i] * vec_rhs[ip];
                vec_rhs[i] = p / vec_d[i];
                i += stride[(tid >> 5) * (norder + 1) + iorder];
            }
        }
    }
    
}







